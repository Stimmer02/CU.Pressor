#include "hip/hip_runtime.h"
#include "Compressor.h"


__global__ void volumeControl(float* data, int size, float volume){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        data[idx] *= volume;
    }
}

__global__ void cuPressor(hipfftReal* data, int size, float factor, float volume){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        data[idx] = copysignf((-1 / ((1 + factor) * abs(data[idx]) + 1) + 1) * (2 + factor) / (1 + factor), data[idx]) * volume;
    }
}
__global__ void cuPressorBath(hipfftReal* data, int size, float factor, float volume, int addressShift){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y; // do not set block y dimension, only grid y dimension
    if (idx < size){
        int index = idy * (addressShift + size) + idx + addressShift;
        data[index] = copysignf((-1 / ((1 + factor) * abs(data[index] * volume) + 1) + 1) * (2 + factor) / (1 + factor), data[index]);
    }
}
__global__ void fftBandSplit(hipfftComplex* input, hipfftComplex* output, int size, int bandWidth){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y; // do not set block y dimension, only grid y dimension
    if (idx < size){
        bool copyBand = ((idx >= (idy * bandWidth)) && (idx < ((idy + 1) * bandWidth))) || (size - 1 == idx);
        float bandMultiplier = 1 * copyBand + 0.001 * !copyBand;
        int index = idx + idy * size;
        output[index].x = input[idx].x * bandMultiplier;
        output[index].y = input[idx].y * bandMultiplier;
    }
}

__global__ void fftBandSplit_smooth(hipfftComplex* input, hipfftComplex* output, int size, int bandWidth){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y; // do not set block y dimension, only grid y dimension
    if (idx < size){
        float copyBand = 1 - 0.5 * cosf(3.14159265358979323846 * (idx - idy * bandWidth) / bandWidth);
        if ((size - 1 == idx) || (idx == 0)){
            copyBand = 1;
        }
        int index = idx + idy * size;
        output[index].x = input[idx].x * copyBand;
        output[index].y = input[idx].y * copyBand;
    }
}

__global__ void bandMerge(hipfftReal* input, hipfftReal* output, int arrShift, int size, int bandCount){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        output[idx] = input[idx + arrShift];
        for (int i = 1; i < bandCount; i++){
            output[idx] += input[i * (arrShift + size) + idx + arrShift];
        }
        output[idx] /= bandCount;
    }
}


Compressor::Compressor(){
    bandCount = 8;
    cufftR2C = 0;
    cufftC2R = 0;
    workBuffer = new CuShiftBuffer<hipfftReal>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    cufftOutput = CuBufferFactory::createBuffer<hipfftComplex>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    cufftBands = CuBufferFactory::createBuffer<hipfftComplex>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    bands = CuBufferFactory::createBuffer<hipfftReal>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    double temp = 0.4;
    setCompressionFactor1(temp);
    setCompressionFactor2(temp);
    temp = 1;
    setVolume(temp);
    temp = 0.5;
    setPreGain(temp);
    setWindowSize(1024);
}

Compressor::~Compressor(){
    if (cufftR2C != 0) {
        hipfftDestroy(cufftR2C);
    }
    if (cufftC2R != 0) {
        hipfftDestroy(cufftC2R);
    }
    delete cufftOutput;
    delete workBuffer;
    delete cufftBands;
    delete bands;
}

void Compressor::compress(float* samplesIn, float* samplesOut, uint size){
	static const uint blockSize = 256;

    CuShiftBuffer<hipfftReal>& workBuffer = *this->workBuffer;
    ACuBuffer<hipfftComplex>& cufftOutput = *this->cufftOutput;
    ACuBuffer<hipfftComplex>& cufftBands = *this->cufftBands;
    ACuBuffer<hipfftReal>& bands = *this->bands;

    if (size <= windowSize){
        uint gridSizeComplex = (complexWindowSize + blockSize - 1) / blockSize;
        uint gridSizeReal = (size + blockSize - 1) / blockSize;
        dim3 gridSizeReal2D = {gridSizeReal, bandCount};
        dim3 gridSizeComplex2D = {gridSizeComplex, bandCount};

        uint addressShift = windowSize - size;

        
        workBuffer.pushBack(FROM_HOST, samplesIn, size);

        volumeControl<<<gridSizeReal, blockSize>>>(workBuffer[addressShift], size, preGain * 2);

        hipfftExecR2C(cufftR2C, workBuffer, cufftOutput);
        fftBandSplit<<<gridSizeComplex2D, blockSize>>>(cufftOutput, cufftBands, complexWindowSize, (complexWindowSize - 1) / bandCount);
        hipfftExecC2R(cufftC2R, cufftBands, bands);

        cuPressorBath<<<gridSizeReal2D, blockSize>>>(bands, size, compressionFactor1, (float)bandCount / windowSize, addressShift);
        bandMerge<<<gridSizeReal2D, blockSize>>>(bands, workBuffer.getInactiveBuffer(), addressShift, size, bandCount);

        volumeControl<<<gridSizeReal, blockSize>>>(workBuffer.getInactiveBuffer(), size, volume / (preGain * 2));
        // cuPressor<<<gridSizeReal, blockSize>>>(workBuffer.getInactiveBuffer(), size, compressionFactor2, volume / (preGain * 2));

        workBuffer.copyInactiveBuffer(TO_HOST, samplesOut, size);
    }  else {
        int leftToProcess = size;
        for (int i = 0; i < size - windowSize; i += windowSize){
            compress(samplesIn + i, samplesOut + i, (leftToProcess > windowSize) ? windowSize : leftToProcess);
            leftToProcess -= windowSize;
        }
    }
	hipDeviceSynchronize();
}

void Compressor::resize(uint size){
    workBuffer->resize(size);
    workBuffer->clear();
    cufftOutput->resize(size / 2 + 1);
    cufftBands->resize((size / 2 + 1) * bandCount);
    bands->resize(size * bandCount);
}

void Compressor::setWindowSize(uint size){
    windowSize = size;
    complexWindowSize = windowSize / 2 + 1;
    resize(windowSize);
    if (cufftR2C != 0) {
        hipfftDestroy(cufftR2C);
    }
    if (cufftC2R != 0) {
        hipfftDestroy(cufftC2R);
    }
    hipfftPlan1d(&cufftR2C, windowSize, HIPFFT_R2C, 1);
    hipfftPlan1d(&cufftC2R, windowSize, HIPFFT_C2R, bandCount);
}

void Compressor::setBandCount(uint count){
    if (bandCount == count){
        return;
    }
    cufftBands->resize(complexWindowSize * bandCount);
    bands->resize(windowSize * bandCount);
    bandCount = count;
}

void Compressor::setSampleRate(uint rate){
    sampleRate = rate;
}

void Compressor::setCompressionFactor1(double& parameter){
    static const float minValue = 0.001;
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    double out = parameter * 1.2;
    double multiplier = out;
    for (int i = 0; i < 2; i++){
        out *= multiplier;
    }
    compressionFactor1 = out - (1 - minValue);
}

void Compressor::setCompressionFactor2(double& parameter){
    static const float minValue = 0.001;
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    double out = parameter * 1.2;
    double multiplier = out;
    for (int i = 0; i < 2; i++){
        out *= multiplier;
    }
    compressionFactor2 = out - (1 - minValue);
}

void Compressor::setVolume(double& parameter){
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    volume = parameter;
}

void Compressor::setPreGain(double& parameter){
    if (parameter <= 0){
        parameter = 0.001;
    } else if (parameter > 1){
        parameter = 1;
    }
    preGain = parameter;
}