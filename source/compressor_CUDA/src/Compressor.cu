#include "hip/hip_runtime.h"
#include "Compressor.h"
#include <stdio.h>


__global__ void volumeControl(float* data, int size, float volume){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        data[idx] *= volume;
    }
}

__global__ void cuPressor(hipfftReal* data, int size, float factor, float volume){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        data[idx] = copysignf((-1 / ((1 + factor) * abs(data[idx]) + 1) + 1) * (2 + factor) / (1 + factor), data[idx]) * volume;
    }
}

__global__ void cuPressorBath(hipfftReal* data, int size, float factor, float volume, int addressShift){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y; // do not set block y dimension, only grid y dimension
    if (idx < size){
        int index = idy * (addressShift + size) + idx + addressShift;
        data[index] = copysignf((-1 / ((1 + factor) * abs(data[index] * volume) + 1) + 1) * (2 + factor) / (1 + factor), data[index]);
    }
}

__global__ void fftBandSplit(hipfftComplex* input, hipfftComplex* output, float* bandMasks, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y; // do not set block y dimension, only grid y dimension
    if (idx < size){
        int index = idx + idy * size;
        output[index].x = input[idx].x * bandMasks[index];
        output[index].y = input[idx].y * bandMasks[index];
    }
}

__global__ void bandMerge(hipfftReal* input, hipfftReal* output, int arrShift, int size, int bandCount){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        output[idx] = input[idx + arrShift];
        for (int i = 1; i < bandCount; i++){
            output[idx] += input[i * (arrShift + size) + idx + arrShift];
        }
        output[idx] /= 2.2;
    }
}

Compressor::Compressor(){
    sampleRate = 44100;
    bandCount = 16;
    cufftR2C = 0;
    cufftC2R = 0;
    workBuffer = new CuShiftBuffer<hipfftReal>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    cufftOutput = CuBufferFactory::createBuffer<hipfftComplex>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    cufftBands = CuBufferFactory::createBuffer<hipfftComplex>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    bands = CuBufferFactory::createBuffer<hipfftReal>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    bandMasks = CuBufferFactory::createBuffer<float>(0, CuBufferFactory::bufferType::TIME_OPTIMAL);
    double temp = 0.4;
    setCompressionFactor1(temp);
    setCompressionFactor2(temp);
    temp = 1;
    setVolume(temp);
    temp = 0.5;
    setPreGain(temp);
    setWindowSize(1024*4);
    generateBandMasks();
}

Compressor::~Compressor(){
    if (cufftR2C != 0) {
        hipfftDestroy(cufftR2C);
    }
    if (cufftC2R != 0) {
        hipfftDestroy(cufftC2R);
    }
    delete cufftOutput;
    delete workBuffer;
    delete cufftBands;
    delete bands;
}

void Compressor::compress(float* samplesIn, float* samplesOut, uint size){
	static const uint blockSize = 256;

    CuShiftBuffer<hipfftReal>& workBuffer = *this->workBuffer;
    ACuBuffer<hipfftComplex>& cufftOutput = *this->cufftOutput;
    ACuBuffer<hipfftComplex>& cufftBands = *this->cufftBands;
    ACuBuffer<hipfftReal>& bands = *this->bands;
    ACuBuffer<float>& bandMasks = *this->bandMasks;

    if (size <= windowSize){
        uint gridSizeComplex = (complexWindowSize + blockSize - 1) / blockSize;
        uint gridSizeReal = (size + blockSize - 1) / blockSize;
        dim3 gridSizeReal2D = {gridSizeReal, bandCount};
        dim3 gridSizeComplex2D = {gridSizeComplex, bandCount};

        uint addressShift = windowSize - size;

        
        workBuffer.pushBack(FROM_HOST, samplesIn, size);

        volumeControl<<<gridSizeReal, blockSize>>>(workBuffer[addressShift], size, preGain * 2);

        hipfftExecR2C(cufftR2C, workBuffer, cufftOutput);
        fftBandSplit<<<gridSizeComplex2D, blockSize>>>(cufftOutput, cufftBands, bandMasks, complexWindowSize);
        hipfftExecC2R(cufftC2R, cufftBands, bands);

        cuPressorBath<<<gridSizeReal2D, blockSize>>>(bands, size, compressionFactor1, 1.0 / windowSize, addressShift);
        bandMerge<<<gridSizeReal, blockSize>>>(bands, workBuffer.getInactiveBuffer(), addressShift, size, bandCount);

        volumeControl<<<gridSizeReal, blockSize>>>(workBuffer.getInactiveBuffer(), size, volume / (preGain * 2));
        cuPressor<<<gridSizeReal, blockSize>>>(workBuffer.getInactiveBuffer(), size, compressionFactor2, volume / (preGain * 2));

        workBuffer.copyInactiveBuffer(TO_HOST, samplesOut, size);
    } else {
        int leftToProcess = size;
        for (int i = 0; i < size - windowSize; i += windowSize){
            compress(samplesIn + i, samplesOut + i, (leftToProcess > windowSize) ? windowSize : leftToProcess);
            leftToProcess -= windowSize;
        }
    }
	hipDeviceSynchronize();
}

void Compressor::resize(uint size){
    workBuffer->resize(size);
    workBuffer->clear();
    cufftOutput->resize(size / 2 + 1);
    cufftBands->resize((size / 2 + 1) * bandCount);
    bands->resize(size * bandCount);
    bandMasks->resize(bandCount + 1);
}

void Compressor::setWindowSize(uint size){
    windowSize = size;
    complexWindowSize = windowSize / 2 + 1;
    resize(windowSize);
    if (cufftR2C != 0) {
        hipfftDestroy(cufftR2C);
    }
    if (cufftC2R != 0) {
        hipfftDestroy(cufftC2R);
    }
    hipfftPlan1d(&cufftR2C, windowSize, HIPFFT_R2C, 1);
    hipfftPlan1d(&cufftC2R, windowSize, HIPFFT_C2R, bandCount);
}

void Compressor::setBandCount(uint count){
    if (bandCount == count){
        return;
    }
    cufftBands->resize(complexWindowSize * bandCount);
    bands->resize(windowSize * bandCount);
    bandCount = count;
    generateBandMasks();
}

void Compressor::setSampleRate(uint rate){
    sampleRate = rate;
    generateBandMasks();
}

float softSignDescend(float x, const float& minVal, const float& range){
    x = -8*(x/range - 0.5);
    return (x/(1.6*(1 + abs(x))) + 0.5) * (1 - minVal) + minVal;
}

float linearDescend(const float& x, const float& minVal, const float& range){
   return 1 - x / range * (1 - minVal);
}

void Compressor::generateBandMasks(){
    float* masks = new float[bandCount * complexWindowSize];
    float maxFrequency = 25000.0;
    float minFrequency = 20.0;
    float sizeRatio = std::pow(maxFrequency / minFrequency, 1.0 / bandCount);
    float maskMinValue = 0.05;
    float previousbandHalfWidth = 0;
    for (int i = 0; i < bandCount; i++){
        float startFrequency = minFrequency * std::pow(sizeRatio, i);
        float endFrequency = minFrequency * std::pow(sizeRatio, i + 1);
        float bandHalfWidth = (endFrequency - startFrequency) / 2;
        // printf("Band %d: %.04f - %.04f\n", i, startFrequency, endFrequency);
    
        for (int j = 1; j < complexWindowSize - 1; j++){
            float frequency = (float)j / (float)complexWindowSize * (float)sampleRate;
            if (frequency < startFrequency + previousbandHalfWidth){ //TODO: check which condition will be more frequent
                float valueBelow = startFrequency + previousbandHalfWidth - frequency;
                if (valueBelow > previousbandHalfWidth * 2){
                    masks[i * complexWindowSize + j] = maskMinValue;
                } else {
                    masks[i * complexWindowSize + j] = softSignDescend(valueBelow, maskMinValue, previousbandHalfWidth * 2);
                }

            } else if (frequency > endFrequency - bandHalfWidth){
                float valueAbove = frequency - (endFrequency - bandHalfWidth);
                if (valueAbove > bandHalfWidth * 2){
                    masks[i * complexWindowSize + j] = maskMinValue;
                } else {
                    masks[i * complexWindowSize + j] = softSignDescend(valueAbove, maskMinValue, bandHalfWidth * 2);
                }

            } else {
                masks[i * complexWindowSize + j] = 1;
            }
            // if (frequency > startFrequency - previousbandHalfWidth && frequency < endFrequency + bandHalfWidth){
            //     printf("   %.04f, %.04f\n", frequency, masks[i * complexWindowSize + j]);
            // }
        }
        masks[i * complexWindowSize] = 1;
        masks[i * complexWindowSize + complexWindowSize - 1] = 1;
        previousbandHalfWidth = bandHalfWidth;
    }

    bandMasks->copyBuffer(FROM_HOST, masks, bandCount * complexWindowSize);
    delete[] masks;
}

void Compressor::setCompressionFactor1(double& parameter){
    static const float minValue = 0.001;
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    double out = parameter * 2; // TODO: find a volume corelation 
    double multiplier = out;
    for (int i = 0; i < 2; i++){
        out *= multiplier;
    }
    compressionFactor1 = out - (1 - minValue);
}

void Compressor::setCompressionFactor2(double& parameter){
    static const float minValue = 0.001;
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    double out = parameter * 1.2;
    double multiplier = out;
    for (int i = 0; i < 2; i++){
        out *= multiplier;
    }
    compressionFactor2 = out - (1 - minValue);
}

void Compressor::setVolume(double& parameter){
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    volume = parameter;
}

void Compressor::setPreGain(double& parameter){
    if (parameter <= 0){
        parameter = 0.001;
    } else if (parameter > 1){
        parameter = 1;
    }
    preGain = parameter;
}