#include "hip/hip_runtime.h"
#include "Compressor.h"


__global__ void cuPressor(float* data, int size, float factor, float volume){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size){
        data[idx] = copysignf((-1 / ((1 + factor) * abs(data[idx]) + 1) + 1) * (2 + factor) / (1 + factor), data[idx]) * volume;
    }
}

__global__ void cuPressorComplex1(hipfftComplex* data, int size, float factor, int windowSize){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size){
        data[idx].x = (-1 / ((1 + factor) * data[idx].x / windowSize + 1) + 1) * (2 + factor) / (1 + factor);
        data[idx].y = (-1 / ((1 + factor) * data[idx].y / windowSize + 1) + 1) * (2 + factor) / (1 + factor);
    }
}

__global__ void cuPressorComplex(hipfftComplex* data, int size, float factor, int windowSize){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size){
        float magnitude = hypotf(data[idx].x, data[idx].y);
        float phase = atan2f(data[idx].y, data[idx].x);
        magnitude = (-1 / ((1 + factor) * magnitude / windowSize + 1) + 1) * (2 + factor) / (1 + factor);
        data[idx].x = magnitude * cosf(phase);
        data[idx].y = magnitude * sinf(phase);
    }
}

__global__ void shiftData(float* inputBuffer, float* outputBuffer, int size, int shift){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // For optimal thread usage, max idx should be size - shift
    if (idx < size - shift){
        outputBuffer[idx] = inputBuffer[idx + shift];
    }
}

Compressor::Compressor(){
    cufftR2C = 0;
    cufftC2R = 0;
    allocatedMemorySize = 0;
    workBufferIndex = 0;
    d_workBuffer[0] = nullptr;
    d_workBuffer[1] = nullptr;
    d_cufftOutput = nullptr;
    compressionFactor1 = 0.5;
    setWindowSize(1024);
}

Compressor::~Compressor(){
    if (cufftR2C != 0) {
        hipfftDestroy(cufftR2C);
    }
    if (cufftC2R != 0) {
        hipfftDestroy(cufftC2R);
    }
    if (d_workBuffer[0] != nullptr) {
        hipFree(d_workBuffer[0]);
    }
    if (d_workBuffer[1] != nullptr) {
        hipFree(d_workBuffer[1]);
    }
    if (d_cufftOutput != nullptr) {
        hipFree(d_cufftOutput);
    }
}

void Compressor::compress(float* samplesIn, float* samplesOut, int size){
	static const int blockSize = 256;

    if (size < windowSize){
        int numBlocksFragment = (windowSize - size + blockSize - 1) / blockSize;
        shiftData<<<numBlocksFragment, blockSize>>>(d_workBuffer[workBufferIndex], d_workBuffer[!workBufferIndex], windowSize, size);
        workBufferIndex = !workBufferIndex;
        hipMemcpy(d_workBuffer[workBufferIndex] + windowSize - size, samplesIn, size * sizeof(float), hipMemcpyHostToDevice);
        hipfftExecR2C(cufftR2C, d_workBuffer[workBufferIndex], d_cufftOutput);
        int numBlocksComplex = (windowSize / 2 + blockSize) / blockSize;
        cuPressorComplex<<<numBlocksComplex, blockSize>>>(d_cufftOutput, windowSize / 2 + 1, compressionFactor1, windowSize);
        hipfftExecC2R(cufftC2R, d_cufftOutput, d_workBuffer[!workBufferIndex]);
        cuPressor<<<numBlocksFragment, blockSize>>>(d_workBuffer[!workBufferIndex] + windowSize - size, size, compressionFactor2, volume);
        hipMemcpy(samplesOut, d_workBuffer[!workBufferIndex] + windowSize - size, size * sizeof(float), hipMemcpyDeviceToHost);
    } else if (size == windowSize){
        hipMemcpy(d_workBuffer[workBufferIndex], samplesIn, size * sizeof(float), hipMemcpyHostToDevice);
        hipfftExecR2C(cufftR2C, d_workBuffer[workBufferIndex], d_cufftOutput);
        int numBlocksComplex = (windowSize / 2 + blockSize) / blockSize;
        cuPressorComplex<<<numBlocksComplex, blockSize>>>(d_cufftOutput, windowSize / 2 + 1, compressionFactor1, windowSize);
        hipfftExecC2R(cufftC2R, d_cufftOutput, d_workBuffer[!workBufferIndex]);
        int numBlocksAll = (windowSize + blockSize - 1) / blockSize;
        cuPressor<<<numBlocksAll, blockSize>>>(d_workBuffer[!workBufferIndex] + windowSize - size, size, compressionFactor2, volume);
        hipMemcpy(samplesOut, d_workBuffer[!workBufferIndex], size * sizeof(float), hipMemcpyDeviceToHost);
    } else {
        int leftToProcess = size;
        for (int i = 0; i < size - windowSize; i += windowSize){
            compress(samplesIn + i, samplesOut + i, (leftToProcess > windowSize) ? windowSize : leftToProcess);
            leftToProcess -= windowSize;
        }
    }
	hipDeviceSynchronize();
}

void Compressor::allocateIfNeeded(int size){
	if (allocatedMemorySize < size){
		if (d_workBuffer[0] != nullptr) {
            hipFree(d_workBuffer[0]);
        }
        if (d_workBuffer[1] != nullptr) {
            hipFree(d_workBuffer[1]);
        }
        if (d_cufftOutput != nullptr) {
            hipFree(d_cufftOutput);
        }
		hipMalloc(&d_workBuffer[0], size * sizeof(hipfftReal));
		hipMalloc(&d_workBuffer[1], size * sizeof(hipfftReal));
        hipMalloc(&d_cufftOutput, (size / 2 + 1) * sizeof(hipfftComplex));

        hipMemset(d_workBuffer[0], 0, size * sizeof(hipfftReal));
        hipMemset(d_workBuffer[1], 0, size * sizeof(hipfftReal));
        hipMemset(d_cufftOutput, 0, (size / 2 + 1) * sizeof(hipfftComplex));
		allocatedMemorySize = size;
	}
}

void Compressor::setWindowSize(int size){
    windowSize = size;
    allocateIfNeeded(windowSize);
    if (cufftR2C != 0) {
        hipfftDestroy(cufftR2C);
    }
    if (cufftC2R != 0) {
        hipfftDestroy(cufftC2R);
    }
    hipfftPlan1d(&cufftR2C, windowSize, HIPFFT_R2C, 1);
    hipfftPlan1d(&cufftC2R, windowSize, HIPFFT_C2R, 1);
}

void Compressor::setCompressionFactor1(double& parameter){
    static const float minValue = 0.001;
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    double out = parameter * 1.2;
    double multiplier = out;
    for (int i = 0; i < 2; i++){
        out *= multiplier;
    }
    compressionFactor1 = out - (1 - minValue);
}

void Compressor::setCompressionFactor2(double& parameter){
    static const float minValue = 0.001;
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    double out = parameter * 1.2;
    double multiplier = out;
    for (int i = 0; i < 2; i++){
        out *= multiplier;
    }
    compressionFactor1 = out - (1 - minValue);
}

void Compressor::setVolume(double& parameter){
    if (parameter < 0){
        parameter = 0;
    } else if (parameter > 1){
        parameter = 1;
    }
    volume = parameter;
}