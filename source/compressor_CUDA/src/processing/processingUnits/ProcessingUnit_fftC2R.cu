#include "ProcessingUnit_fftC2R.h"

ProcessingUnit_fftC2R::ProcessingUnit_fftC2R(hipfftComplex*& d_input, hipfftReal*& d_output, hipfftHandle& plan)
    : d_input(d_input), d_output(d_output), plan(plan){}

void ProcessingUnit_fftC2R::process(){
    hipfftExecC2R(plan, d_input, d_output);
}