#include "hip/hip_runtime.h"
#include "ProcessingUnit_cuPressorBath.h"

__global__ void cuPressorBath(float* data, int size, float* factors, float volume, int addressShift){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y; // do not set block y dimension, only grid y dimension
    if (idx < size){
        int index = idy * (addressShift + size) + idx + addressShift;
        data[index] = copysignf((-1 / ((1 + factors[idy]) * abs(data[index] * volume) + 1) + 1) * (2 + factors[idy]) / (1 + factors[idy]), data[index]);
    }
}

ProcessingUnit_cuPressorBath::ProcessingUnit_cuPressorBath(float*& d_workBuffer, const uint& gridSize, const uint& blockSize, const uint& bufferSize, const uint bandCount, const uint& addressShift)
    : d_workBuffer(d_workBuffer), gridSize(gridSize), blockSize(blockSize), bufferSize(bufferSize), addressShift(addressShift), bandCount(bandCount) {
    preGain = 1;
    
    compressionFactors = new float[bandCount];
    for (int i = 0; i < bandCount; i++){
        compressionFactors[i] = 0.4;
    }
    activeFactors = bandCount;

    hipMalloc(&d_compressionFactors, bandCount * sizeof(float));
    hipMemcpy(d_compressionFactors, compressionFactors, bandCount * sizeof(float), hipMemcpyHostToDevice);
}

ProcessingUnit_cuPressorBath::~ProcessingUnit_cuPressorBath(){
    hipFree(d_compressionFactors);
    delete[] compressionFactors;
}

void ProcessingUnit_cuPressorBath::process(){
    cuPressorBath<<<(gridSize, bandCount), blockSize>>>(d_workBuffer, bufferSize, d_compressionFactors, preGain, addressShift);
}

void ProcessingUnit_cuPressorBath::setCompressionFactor(uint bandIndex, float factor){
    static const float minValue = 0.001;

    if (bandIndex >= bandCount){
        return;
    }
    if (compressionFactors[bandIndex] == 0 && factor != 0){
        activeFactors++;
    } else if (compressionFactors[bandIndex] != 0 && factor == 0){
        activeFactors--;
    }
    compressionFactors[bandIndex] = factor;

    setActive(activeFactors != 0);

    double out = factor * 2; // TODO: find a volume corelation 
    double multiplier = out;
    for (int i = 0; i < 2; i++){
        out *= multiplier;
    }
    factor = out - (1 - minValue);

    hipMemcpy(d_compressionFactors + bandIndex, &factor, sizeof(float), hipMemcpyHostToDevice);
}

float ProcessingUnit_cuPressorBath::getCompressionFactor(uint bandIndex) const{
    if (bandIndex >= bandCount){
        return INFINITY;
    }
    return compressionFactors[bandIndex];
}

void ProcessingUnit_cuPressorBath::setPreGain(float gain){
    preGain = gain;
}

float ProcessingUnit_cuPressorBath::getPreGain() const{
    return preGain;
}

