#include "CuBuffer_memoryOptimal.h"

template <typename TYPE>
CuBuffer_memoryOptimal<TYPE>::CuBuffer_memoryOptimal() : ACuBuffer<TYPE>(){}

template <typename TYPE>
CuBuffer_memoryOptimal<TYPE>::CuBuffer_memoryOptimal(uint size) : ACuBuffer<TYPE>(size){}

template <typename TYPE>
CuBuffer_memoryOptimal<TYPE>::CuBuffer_memoryOptimal(CuBuffer_memoryOptimal*& cuBuffer) : ACuBuffer<TYPE>(cuBuffer){}

template <typename TYPE>
void CuBuffer_memoryOptimal<TYPE>::resize(uint size){
    if (this->size == size){
        return;
    }

    TYPE* d_newBuffer, d_oldBuffer = d_buffer;
    hipMalloc(&d_newBuffer, size * sizeof(TYPE));
    if (size < this->size){
        hipMemcpy(d_newBuffer, d_buffer, size * sizeof(TYPE), hipMemcpyDeviceToDevice);
    } else {
        hipMemcpy(d_newBuffer, d_buffer, this->size * sizeof(TYPE), hipMemcpyDeviceToDevice);
    }
    d_buffer = d_newBuffer;
    this->size = size;
    allocatedSize = size;
    hipFree(d_oldBuffer);
}
