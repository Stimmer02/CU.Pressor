#include "hip/hip_runtime.h"
#include "CuShiftBuffer.h"

template <typename TYPE>
__global__ void kernel_shiftData(TYPE* inputBuffer, TYPE* outputBuffer, int size, int shift){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // For optimal thread usage, max idx should be size - shift
    if (idx < size - shift){
        outputBuffer[idx] = inputBuffer[idx + shift];
    }
}


template <typename TYPE>
CuShiftBuffer<TYPE>::CuShiftBuffer(uint size, typename CuBufferFactory::bufferType bufferType){
    CuBufferFactory::fillBufferArray<TYPE>(buffer, 2, size, bufferType);
    setSize(buffer[0]->getSize());
    setAllocatedSize(buffer[0]->getAllocatedSize());
    activeBuffer = 0;
}

template <typename TYPE>
CuShiftBuffer<TYPE>::~CuShiftBuffer(){
    deallocate();
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::shift(uint shiftSize){
    static const int blockSize = 256;
    int gridSize = (getSize() - shiftSize + blockSize - 1) / blockSize;
    kernel_shiftData<TYPE><<<gridSize, blockSize>>>(buffer[activeBuffer]->getBuffer(), buffer[!activeBuffer]->getBuffer(), getSize(), shiftSize);
    activeBuffer = !activeBuffer;
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::put(originType origin, TYPE* d_arr, uint size){
    if (size >= getSize()){
        hipMemcpy(buffer[activeBuffer]->getBuffer(), d_arr, getSize() * sizeof(TYPE), (hipMemcpyKind)origin);
    } else {
        shift(size);
        hipMemcpy(buffer[activeBuffer]->getBuffer() + buffer[activeBuffer]->getSize() - size, d_arr, size * sizeof(TYPE), (hipMemcpyKind)origin); 
    }
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::setBuffer(ACuBuffer<TYPE>*& cuBuffer){
    buffer[activeBuffer]->deallocate();
    buffer[!activeBuffer]->allocate(cuBuffer->getAllocatedSize());

    setSize(cuBuffer->getSize());
    setAllocatedSize(cuBuffer->getAllocatedSize());
    buffer[activeBuffer]->setBuffer(cuBuffer);
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::setBuffer(TYPE*& d_buffer, uint size, uint allocatedSize){
    allocatedSize = allocatedSize != 0 ? allocatedSize : size;
    buffer[activeBuffer]->deallocate();
    buffer[!activeBuffer]->allocate(allocatedSize);

    setSize(size);
    setAllocatedSize(allocatedSize);
    buffer[activeBuffer]->setBuffer(d_buffer, size, allocatedSize);
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::resize(uint newSize){
    buffer[0]->resize(newSize);
    buffer[1]->resize(newSize);
    setSize(buffer[0]->getSize());
    setAllocatedSize(buffer[0]->getAllocatedSize());
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::deallocate(){
    buffer[0]->deallocate();
    buffer[1]->deallocate();
    setSize(0);
    setAllocatedSize(0);
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::allocate(uint size){
    buffer[0]->allocate(size);
    buffer[1]->allocate(size);
    setSize(size);
    setAllocatedSize(size);
}

template <typename TYPE>
uint CuShiftBuffer<TYPE>::getSize() const {
    return size;
}

template <typename TYPE>
uint CuShiftBuffer<TYPE>::getAllocatedSize() const {
    return allocatedSize;
}

template <typename TYPE>
TYPE* CuShiftBuffer<TYPE>::getBuffer() const {
    return buffer[activeBuffer]->getBuffer();
}

template <typename TYPE>
TYPE* CuShiftBuffer<TYPE>::getInactiveBuffer(uint index) const {
    return buffer[!activeBuffer]->getBuffer() + index;
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::copyInactiveBuffer(destinationType destination, TYPE* d_buffer, uint size, uint index) const {
    buffer[!activeBuffer]->copyBuffer(destination, d_buffer, size, index);
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::setSize(uint size){
    this->size = size;
    buffer[0]->setSize(size);
    buffer[1]->setSize(size);
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::setAllocatedSize(uint allocatedSize){
    this->allocatedSize = allocatedSize;
    buffer[0]->setAllocatedSize(allocatedSize);
    buffer[1]->setAllocatedSize(allocatedSize);
}

template <typename TYPE>
void CuShiftBuffer<TYPE>::setBufferToNull(){
    buffer[0]->setBufferToNull();
    buffer[1]->setBufferToNull();
}

template <typename TYPE>
TYPE* CuShiftBuffer<TYPE>::getBufferOvnership(){
    TYPE* d_buffer = buffer[activeBuffer]->getBuffer();
    buffer[activeBuffer]->setBufferToNull();
    deallocate();
    return d_buffer;
}